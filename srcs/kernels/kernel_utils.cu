#include "hip/hip_runtime.h"
#include "kernel_utils.h"

__global__ void d_reduction(float* matrix, int height, int width, float* result)
{
  extern __shared__ float thread_sums[];
  int offset = blockIdx.x * blockDim.x + threadIdx.x;

  float local_sum = 0;
  for (int i = 0; i < height / blockDim.x + 1; ++i) {
    if (offset < height)
      local_sum += matrix[offset];
    offset += blockDim.x;
  }
  __syncthreads();

  if (blockDim.x > 512 && threadIdx.x < 512)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 512];
  __syncthreads();
  if (blockDim.x > 256 && threadIdx.x < 256)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 256];
  __syncthreads();
  if (blockDim.x > 128 && threadIdx.x < 128)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 128];
  __syncthreads();
  if (blockDim.x > 64 && threadIdx.x < 64)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 64];
  __syncthreads();
  if (blockDim.x > 32 && threadIdx.x < 32)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 32];
  __syncthreads();
  for (int i = warpSize / 2; i > 0; i /= 2)
    local_sum = __shfl_down_sync(0xFFFFFFFF, local_sum, i);

  if (threadIdx.x == 0)
    result[blockIdx.x] = local_sum;
}

extern "C" void reduction(float* matrix, int height, int width, float* result)
{
  d_reduction<<<width, REDUCTION_THREADS, REDUCTION_THREADS * sizeof(float)>>>(matrix, height, width, result);
}
