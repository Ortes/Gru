#include "hip/hip_runtime.h"
#include "kernel_utils.h"
#include "constant.h"

__global__ void d_exp_reduction(float* matrix, float* exp_matrix, int height, int width, float* result)
{
  extern __shared__ float thread_sums[];

  int offset;
  if (!(height % 32))
    offset = blockIdx.x * height + threadIdx.x;
  else
    offset = blockIdx.x * ALIGN32(height) + threadIdx.x;

  float local_sum = 0;
  for (int i = 0; i < height / blockDim.x + 1 && offset < height; ++i, offset += blockDim.x)
    local_sum += exp_matrix[offset] = EXP_FUNC(matrix[offset]);

  thread_sums[threadIdx.x] = local_sum;
  __syncthreads();

  if (blockDim.x > 512 && threadIdx.x < 512)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 512];
  __syncthreads();
  if (blockDim.x > 256 && threadIdx.x < 256)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 256];
  __syncthreads();
  if (blockDim.x > 128 && threadIdx.x < 128)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 128];
  __syncthreads();
  if (blockDim.x > 64 && threadIdx.x < 64)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 64];
  __syncthreads();
  if (blockDim.x > 32 && threadIdx.x < 32)
    thread_sums[threadIdx.x] = local_sum = local_sum + thread_sums[threadIdx.x + 32];
  __syncthreads();

  for (int i = warpSize / 2; i > 0; i /= 2)
    local_sum = __shfl_down_sync(0xFFFFFFFF, local_sum, i);

  if (threadIdx.x == 0)
    result[blockIdx.x] = local_sum;
}

__global__ void d_exp(float* array, int size)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
    array[index] = exp(array[index]);
}

__global__ void d_softmax(float* matrix, int height, int width, float* exp_sum)
{
  int size = ALIGN32(height) * width;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
    matrix[index] /= exp_sum[index / ALIGN32(height)];
}
__global__ void d_w2o_update(float* weights, int height, int width, int batch_size, float* exp_vec, float* ouput, float* expected)
{
  int size = ALIGN32(height) * width;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
    ;
}


// cpu call wrappers

extern "C" void m_exp_reduction(float* matrix, float* exp_matrix, int height, int width, float* result)
{
  d_exp_reduction<<<width, REDUCTION_THREADS, REDUCTION_THREADS * sizeof(float)>>>(matrix, exp_matrix, height, width, result);
}

extern "C" void v_exp(float* array, int size)
{
  d_exp<<<size / EXP_THREADS + 1, EXP_THREADS>>>(array, size);
}

extern "C" void m_softmax(float* matrix, int height, int width, float* exp_sum)
{
  d_softmax<<<(ALIGN32(height) * width) / SOFTMAX_THREADS + 1, SOFTMAX_THREADS>>>(matrix, height, width, exp_sum);
}

extern "C" void w2o_update(float* weights, int height, int width, int batch_size, float* exp_vec, float* ouput, float* expected)
{
  d_w2o_update<<<(ALIGN32(height) * width) / UPDATE_L1_THREADS + 1, UPDATE_L1_THREADS>>>(weights, height, width, batch_size, exp_vec, ouput, expected);
}
