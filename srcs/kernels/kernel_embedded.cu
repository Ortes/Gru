#include "hip/hip_runtime.h"
__global__ void reduction(float* matrix, int height, int width)
{
  __shared__ float thread_sums[];
  int offset = blockIdx.x * blockDim.x + threadIdx.x;

  float local_sum = 0;
  for (int i = ; i < height / blockDim.x + 1; ++i) {
    if (offset < height)
      local_sum += matrix[offset];
    offset += blockDim.x;
  }

  if (blockDim.x > 512 && threadIdx.x < 512)
    thread_sums[threadIdx.x] = localSum = local_sum + thread_sums[threadIdx.x + 512];
  if (blockDim.x > 256 && threadIdx.x < 256)
    thread_sums[threadIdx.x] = localSum = local_sum + thread_sums[threadIdx.x + 256];
  if (blockDim.x > 128 && threadIdx.x < 128)
    thread_sums[threadIdx.x] = localSum = local_sum + thread_sums[threadIdx.x + 128];
  if (blockDim.x > 64 && threadIdx.x < 64)
    thread_sums[threadIdx.x] = localSum = local_sum + thread_sums[threadIdx.x + 64];
  if (blockDim.x > 32 && threadIdx.x < 32)
    thread_sums[threadIdx.x] = localSum = local_sum + thread_sums[threadIdx.x + 32];

  for (int i = warpSize / 2; i > 0; i /= 2)
    mySum = __shfl_down_sync(mySum, i);

  if (threadIdx.x = 0)
    return localSum;
}
